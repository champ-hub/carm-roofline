#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

// DEFINE PRECISION

__global__ void benchmark(PRECISION *d_X, int iterations) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// DEFINE INITIALIZATION

	for (int i = 0; i < iterations; i++) {
#pragma unroll
		for (int j = 0; j < 128; j++) {
			// DEFINE LOOP
		}
	}

	d_X[id] = d;
}