#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdlib.h>

#include <algorithm>
#include <iostream>
#include <vector>

using namespace std;

// DEFINE NUM_REPS

// DEFINE KERNEL PARAMETERS

#define A_SIZE M *K *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define B_SIZE K *N *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define C_SIZE M *N *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS

// DEFINE PRECISION

// DEFINE DEVICE

__global__ void benchmark(PRECISION_A *d_A, PRECISION_B *d_B, PRECISION_C *d_C, int iterations) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// DEFINE INITIALIZATION

	for (int i = 0; i < iterations; i++) {
#pragma unroll
		for (int j = 0; j < 128; j++) {
			// DEFINE LOOP
		}
	}
	d_C[id] = fragsC[0];
}

int main() {
	// Allocate memory in GPU
	hipSetDevice(DEVICE);

	int iterations = 1;
	float milliseconds = 0;
	vector<float> time_series;

	PRECISION_A *d_A;
	hipMalloc((void **)&d_A, A_SIZE * sizeof(PRECISION_A));
	PRECISION_B *d_B;
	hipMalloc((void **)&d_B, B_SIZE * sizeof(PRECISION_B));
	PRECISION_C *d_C;
	hipMalloc((void **)&d_C, C_SIZE * sizeof(PRECISION_C));

	// Timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Determine the minimum number of iterations
	while (milliseconds < 150.f) {
		iterations *= 2;
		hipEventRecord(start);

		benchmark<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, iterations);
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&milliseconds, start, stop);
	}

	// Perform testing
	for (int i = 0; i < NUM_REPS; i++) {
		hipEventRecord(start);

		benchmark<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, iterations);
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&milliseconds, start, stop);
		time_series.push_back(milliseconds);
	}

	// calculate median of execution time
	sort(time_series.begin(), time_series.end());
	float median = 0;

	if (time_series.size() % 2 == 0) {
		median =
			(time_series[time_series.size() / 2] + time_series[time_series.size() / 2 - 1]) / 2;
	} else {
		median = time_series[time_series.size() / 2];
	}

	double flops = 2. * M * N * K * iterations * 128 * (THREADS_PER_BLOCK / 32.) * NUM_BLOCKS / 1e9;
	float perf = flops * 1e3 / median;

	cout << perf << " GFLOPS/s" << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}
