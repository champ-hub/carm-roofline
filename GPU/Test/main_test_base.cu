#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <iostream>

using namespace std;

// DEFINE KERNEL PARAMETERS

// DEFINE PRECISION

// DEFINE DEVICE

__global__ void benchmark(float *d_X);

int main() {
	// Allocate memory in GPU
	hipSetDevice(DEVICE);
	PRECISION *h_X = (PRECISION *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(PRECISION));
	PRECISION *d_X;
	hipMalloc((void **)&d_X, NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(PRECISION));

	// Timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	benchmark<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_X);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipMemcpy(h_X, d_X, NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(PRECISION),
			   hipMemcpyDeviceToHost);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	double flops = 2. * 4 * ITERATIONS * THREADS_PER_BLOCK * NUM_BLOCKS / 1e9;
	float perf = flops * 1e3 / milliseconds;

	cout << perf << " GFLOPS" << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_X);
	return 0;
}