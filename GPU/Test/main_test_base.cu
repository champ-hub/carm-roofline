#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>

using namespace::std;

//DEFINE KERNEL PARAMETERS

//DEFINE PRECISION

//DEFINE DEVICE

__global__ void benchmark(float *d_X);

int main() {
	// Allocate memory in GPU
	hipSetDevice(DEVICE);
	PRECISION *d_X;
	hipMalloc((void **)&d_X, NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(PRECISION));

	// Timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	benchmark<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_X);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	cout << milliseconds << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_X);
	return 0;
}