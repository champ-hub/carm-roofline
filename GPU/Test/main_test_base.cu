#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <algorithm>
#include <iostream>
#include <vector>

using namespace std;

#define NUM_REPS 1024

// DEFINE KERNEL PARAMETERS

// DEFINE PRECISION

// DEFINE DEVICE

__global__ void benchmark(PRECISION *d_X, int iterations);

int main() {
	// Allocate memory in GPU
	hipSetDevice(DEVICE);

	int iterations = 10;
	float milliseconds = 0;
	vector<float> time_series;

	PRECISION *d_X;
	hipMalloc((void **)&d_X, NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(PRECISION));

	// Timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Determine the minimum number of iterations
	while (milliseconds < 150.f) {
		iterations *= 2;
		hipEventRecord(start);
		benchmark<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_X, iterations);
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&milliseconds, start, stop);
	}

	// Perform testing
	for (int i = 0; i < NUM_REPS; i++) {
		hipEventRecord(start);
		benchmark<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_X, iterations);
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&milliseconds, start, stop);
		time_series.push_back(milliseconds);
	}

	// calculate median of execution time
	sort(time_series.begin(), time_series.end());
	float median = 0;

	if (time_series.size() % 2 == 0) {
		median =
			(time_series[time_series.size() / 2] + time_series[time_series.size() / 2 - 1]) / 2;
	} else {
		median = time_series[time_series.size() / 2];
	}

	double flops = 2. * 4 * iterations * 128 * THREADS_PER_BLOCK * NUM_BLOCKS / 1e9;
	float perf = flops * 1e3 / median;

	cout << perf << " GFLOPS" << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_X);
	return 0;
}